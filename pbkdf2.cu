#include "hip/hip_runtime.h"
#include "hmac-sha1.cu"
#include <stdio.h>
#include "/usr/local/cuda-sdk/C/common/inc/cutil.h"
#include <errno.h>

#define PASS 8 //needs to be devisible by 4. if not then pad
#define HLEN 20
__device__ void ThreadToPass(unsigned int num, char * pass) {
  //assume we just work with numbers
  //we know we have max 9 numbers because of the size of int
 /* int temp = num % 10;
  pass[7] = (char) temp + 48;
  temp = (num % 100) / 10; //remember C always rounds down
  pass[6] = (char) temp + 48;
  temp = (num % 1000) / 100;
  pass[5] = (char) temp + 48;
  temp = (num % 10000) / 1000;
  pass[4] = (char) temp + 48;
  temp = (num % 100000) / 10000;
  pass[3] = (char) temp + 48;
  temp = (num % 1000000) / 100000;
  pass[2] = (char) temp + 48;
  temp = (num % 10000000) / 1000000;
  pass[1] = (char) temp + 48;
  temp = (num % 100000000) / 10000000;
  pass[0] = (char) temp + 48;

  //remove the leasing zeros, equivilent to trim
  int zeros = 0;
  for (int i = 0; i <= 7 && pass[i] == '0'; i++) {
    zeros++;
  }
  for (int i = 0; i <=7; i++) {
    if (i >= 8 - zeros) //set null chars
      if (i == 0)
	pass[i] = '0'; //if we try to encode the number 0
      else
	pass[i] = '\0';
    else
      pass[i] = pass[zeros+i];
  }*/
   pass[7]='w';
   pass[6]='o';
   pass[5]='r';
   pass[4]='d';
   pass[3]='d';
   pass[2]='d';
   pass[1]='d';
   pass[0]='d';
}

__device__ void F (size_t hLen, 
		   const void * passwordPtr, size_t passwordLen,
		   const void * saltPtr, size_t saltLen,
		   size_t iterationCount,
		   size_t blockNumber,
		   void *dataPtr, size_t dkLen,
		   void *tempBuffer,
		   struct globalChars *chars)
{

  uint8_t *inBlock, *outBlock, *resultBlockPtr;
  size_t iteration;
  outBlock = (uint8_t*) tempBuffer;
  inBlock = outBlock + hLen;
  /* Set up inBlock to contain Salt || INT (blockNumber). */
  cudaMemcpyDevice(inBlock, saltPtr, saltLen);
		
  ((uint32_t *) inBlock)[saltLen/4] = SWAP(blockNumber);

  // Caculate U1 (result goes to outBlock) and copy it to resultBlockPtr. 
  resultBlockPtr = (uint8_t*) dataPtr;
  hmac_sha1 (passwordPtr, passwordLen, inBlock, saltLen + 4, outBlock, chars);
  cudaMemcpyDevice(resultBlockPtr, outBlock, hLen);
  // Calculate U2 though UiterationCount. 
  for (iteration = 2; iteration <= iterationCount; iteration++)
    {
      uint8_t *tempBlock;
      size_t byte;
      // Swap inBlock and outBlock pointers. 
      tempBlock = inBlock; 
      inBlock = outBlock;
      outBlock = tempBlock;
      // Now inBlock conatins Uiteration-1.  Calclulate Uiteration into outBlock. 
      hmac_sha1 (passwordPtr, passwordLen, inBlock, hLen, outBlock, chars);
      // Xor data in dataPtr (U1 \xor U2 \xor ... \xor Uiteration-1) with
      // outBlock (Uiteration). 
      for (byte = 0; byte < hLen/4; byte++) {
	((uint32_t*)resultBlockPtr)[byte] ^= ((uint32_t*)outBlock)[byte];
      }
    }
}

__global__ void pbkdf2 (size_t passwordLen,
			const void * saltPtr, size_t saltLen,
			size_t iterationCount,
			uint8_t* dkPtr,
			size_t dkLen)
{
  __shared__ uint8_t sharedData[BLOCK_DIM][HLEN];
  __shared__ uint8_t sharedTemp[BLOCK_DIM][2*HLEN];
  __shared__ uint8_t sharedPass[BLOCK_DIM][PASS];
  __shared__ struct globalChars sharedChars[BLOCK_DIM];

  int num = blockIdx.x*blockDim.x + threadIdx.x;
  ThreadToPass(num, (char*)sharedPass[threadIdx.x]);
  __syncthreads();

  size_t completeBlocks;
  //completeBlocks = dkLen / HLEN;
  completeBlocks=dkLen;
  size_t partialBlockSize;
  partialBlockSize = dkLen % HLEN;

  if (completeBlocks == 1) {
    F (HLEN, sharedPass[threadIdx.x], passwordLen, (uint8_t*)saltPtr, saltLen,
       iterationCount, 1, sharedData[threadIdx.x], dkLen,sharedTemp[threadIdx.x], &sharedChars[threadIdx.x]);
  } else {
    // Handle handle the case if partialBlockSize > 0 or if the password is longer than 20 bytes
  }
  __syncthreads();
  size_t gridSize = (size_t) ceil((float)TEST_SIZE / (float)BLOCK_DIM);
  //final block
  if (blockIdx.x == gridSize-1){
    for (int r = 0; r < HLEN; ++r) {
      dkPtr[BLOCK_DIM*threadIdx.x+r] = sharedData[threadIdx.x][r];
    }
  }
  __syncthreads();
}


extern "C" __host__
void doIt( uint32_t iterationCount) {
  clock_t t0, t1;
  //hLen is length of prf output in bytes, for sha1 it is 160 bit = 20 bytes
  size_t hLen = HLEN;
  size_t passwordLen = PASS;
  //password should be a byte string, ie string consisting of characters of 8 bits
  const unsigned int block_dim_x = BLOCK_DIM;// 64; //max threads pr block
  size_t saltLen = 6;
  // TempBuffer should be of size hLen
  //changed for our SSID
  char * salt = "ATT256"; 
  //Length of result key in bytes, most be less than 2^32 * hLen
  size_t dkLen = 32; //hardcoded in our case
  //dKPtr is where the resultant key is stored, also as a byte string
  //aes = 128, 192, 256 bits, ie 16, 24, 32 bytes
  uint8_t* dkPtr;
  void * saltPtr;
  //Each 20 byte value needs 32 bytes to make cudamemcpy happy
  uint8_t * data= (uint8_t* ) malloc(BLOCK_DIM*32);
  double datetime_diff_ms = 0;

  for (int i =1; i<= 1; i++) {
    //Sync to avoid async timing
    hipDeviceSynchronize();
    t0 = clock();

    CUDA_SAFE_CALL(hipMalloc(&dkPtr, BLOCK_DIM*32));
    //only read, no need to be of size 32
    CUDA_SAFE_CALL(hipMalloc(&saltPtr, saltLen));
    CUDA_SAFE_CALL(hipMemcpy(saltPtr, salt, saltLen, hipMemcpyHostToDevice));
    size_t gridSize = (size_t) ceil((float)TEST_SIZE / (float)block_dim_x);
    dim3 block(block_dim_x, 1, 1);
    dim3 grid(gridSize,1,1);
    //first amount of rows

    pbkdf2<<< gridSize, block>>>(passwordLen, saltPtr, saltLen, iterationCount, dkPtr, dkLen);

    CUDA_SAFE_CALL(hipMemcpy(data, dkPtr, BLOCK_DIM*32, hipMemcpyDeviceToHost));

    hipFree(saltPtr);
    hipFree(dkPtr);

    //Sync to avoid async timing
    hipDeviceSynchronize();
    hipDeviceReset();
    t1 = clock();

    datetime_diff_ms += difftime(t1, t0)  / CLOCKS_PER_SEC;
    printf("time diff is %d\n", datetime_diff_ms);
  }
  
  for (int k =0; k<block_dim_x; k++) {
    for (int j =0; j< hLen ; j++) {
      printf("%x",data[k*block_dim_x+j]);
    }
    printf("\n");
  }
  free(data);
}
int main(void)
{
  printf("starting..\n");
  uint32_t iterations =  (uint32_t)4096;
  doIt(iterations);
}
/**
static size_t correctMalloc(size_t input) {
   size_t temp = input;
      size_t add;
      if (temp % 4 != 0) {
      add = 4 -(temp % 4);
      temp += add;
      } 
      input = temp;
  return input;
}*/

void Check_CUDA_Error(const char *message)
{
  if (errno != 0) {
    printf("C Error: %s: %s\n", message, strerror( errno ));
    //exit(-1);
  }
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"CUDA ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }  
}

__forceinline__ __device__
void cudaCorrectMalloc(size_t *input) {
  /* size_t temp = *input;
     size_t add;
     if (temp % 4 != 0) {
     add = 4-(temp % 4);
     temp += add;
     } 
     *input = temp;*/
}
